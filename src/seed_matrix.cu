#include "hip/hip_runtime.h"
#include <rmd/seed_matrix.cuh>
#include <rmd/texture_memory.cuh>
#include <rmd/helper_vector_types.cuh>

#include "seed_init.cu"
#include "seed_update.cu"
#include "seed_check.cu"
#include "epipolar_match.cu"

#include <stdio.h>

rmd::SeedMatrix::SeedMatrix(
    const size_t &width,
    const size_t &height,
    const PinholeCamera &cam)
  : width_(width)
  , height_(height)
  , ref_img_(width, height)
  , curr_img_(width, height)
  , sum_templ_(width, height)
  , const_templ_denom_(width, height)
  , mu_(width, height)
  , sigma_(width, height)
  , a_(width, height)
  , b_(width, height)
  , convergence_(width, height)
  , epipolar_matches_(width, height)
{
  // Save image details to be uploaded to device memory
  dev_data_.ref_img.set(ref_img_);
  dev_data_.curr_img.set(curr_img_);
  dev_data_.sum_templ.set(sum_templ_);
  dev_data_.const_templ_denom.set(const_templ_denom_);
  dev_data_.mu.set(mu_);
  dev_data_.sigma.set(sigma_);
  dev_data_.a.set(a_);
  dev_data_.b.set(b_);
  dev_data_.convergence.set(convergence_);
  dev_data_.epipolar_matches.set(epipolar_matches_);
  // Save camera parameters
  dev_data_.cam    = cam;
  dev_data_.one_pix_angle = cam.getOnePixAngle();
  dev_data_.width  = width;
  dev_data_.height = height;
  // Kernel configuration
  dim_block_.x = 16;
  dim_block_.y = 16;
  dim_grid_.x = (width  + dim_block_.x - 1) / dim_block_.x;
  dim_grid_.y = (height + dim_block_.y - 1) / dim_block_.y;
}

bool rmd::SeedMatrix::setReferenceImage(
    float *host_ref_img_align_row_maj,
    const SE3<float> &T_curr_world,
    const float &min_depth,
    const float &max_depth)
{
  // Upload reference image to device memory
  ref_img_.setDevData(host_ref_img_align_row_maj);
  // Set scene parameters
  dev_data_.scene.min_depth    = min_depth;
  dev_data_.scene.max_depth    = max_depth;
  dev_data_.scene.avg_depth    = (min_depth+max_depth)/2.0f;
  dev_data_.scene.depth_range  = max_depth - min_depth;
  dev_data_.scene.sigma_sq_max = dev_data_.scene.depth_range * dev_data_.scene.depth_range / 36.0f;
  // Algorithm parameters
  dev_data_.eta_inlier  = 0.7f;
  dev_data_.eta_outlier = 0.05f;
  dev_data_.epsilon     = dev_data_.scene.depth_range / 10000.0f;
  // Copy data to device memory
  dev_data_.setDevData();

  T_world_ref_ = T_curr_world.inv();

  rmd::bindTexture(ref_img_tex, ref_img_);

  rmd::seedInitKernel<<<dim_grid_, dim_block_>>>(dev_data_.dev_ptr);
  hipDeviceSynchronize();
  return true;
}

bool rmd::SeedMatrix::update(
    float *host_curr_img_align_row_maj,
    const SE3<float> &T_curr_world)
{
  const rmd::SE3<float> T_curr_ref = T_curr_world * T_world_ref_;

  // Upload current image to device memory
  curr_img_.setDevData(host_curr_img_align_row_maj);
  // Bind texture memory for the current image
  rmd::bindTexture(curr_img_tex, curr_img_);

  // ... and model parameters
  rmd::bindTexture(mu_tex, mu_);
  rmd::bindTexture(sigma_tex, sigma_);
  rmd::bindTexture(a_tex, a_);
  rmd::bindTexture(b_tex, b_);

  printf("zieta\n");

  // Assest current convergence status
  rmd::seedCheckKernel<<<dim_grid_, dim_block_>>>(dev_data_.dev_ptr);
  rmd::bindTexture(convergence_tex, convergence_);

  hipDeviceSynchronize();
  // Establish epipolar correspondences
  // call epipolar matching kernel
  rmd::seedEpipolarMatch<<<dim_grid_, dim_block_>>>(
                                                    dev_data_.dev_ptr,
                                                    T_curr_ref);
  rmd::bindTexture(epipolar_matches_tex, epipolar_matches_);

  rmd::seedUpdateKernel<<<dim_grid_, dim_block_>>>(
                                                   dev_data_.dev_ptr,
                                                   T_curr_ref.inv());
  return true;
}

void rmd::SeedMatrix::downloadDepthmap(float *host_depthmap_align_row_maj)
{
  mu_.getDevData(host_depthmap_align_row_maj);
}

#ifdef RMD_DEBUG
void rmd::SeedMatrix::downloadDisparity(
    float *host_disp_x_align_row_maj,
    float *host_disp_y_align_row_maj)
{
  float2 disp[640*480];
  epipolar_matches_.getDevData(disp);

  for(size_t y = 0; y<480; ++y)
  {
    for(size_t x = 0; x<640; ++x)
    {
      const size_t i = y*640+x;
      host_disp_x_align_row_maj[i] = disp[i].x;
      host_disp_y_align_row_maj[i] = disp[i].y;
    }
  }
}

void rmd::SeedMatrix::downloadConvergence(
    unsigned char *host_conv_align_row_maj)
{
  unsigned char conv[640*480];
  convergence_.getDevData(conv);
  for(size_t y = 0; y<480; ++y)
  {
    for(size_t x = 0; x<640; ++x)
    {
      const size_t i = y*640+x;
      host_conv_align_row_maj[i] = conv[i];
    }
  }
}
#endif
