#include <rmd/seed_matrix.cuh>
#include <rmd/texture_memory.cuh>
#include <rmd/helper_vector_types.cuh>

#include "seed_init.cu"
#include "seed_check.cu"
#include "epipolar_match.cu"
#include "seed_update.cu"

rmd::SeedMatrix::SeedMatrix(
    const size_t &width,
    const size_t &height,
    const PinholeCamera &cam)
  : width_(width)
  , height_(height)
  , ref_img_(width, height)
  , curr_img_(width, height)
  , sum_templ_(width, height)
  , const_templ_denom_(width, height)
  , mu_(width, height)
  , sigma_(width, height)
  , a_(width, height)
  , b_(width, height)
  , convergence_(width, height)
  , epipolar_matches_(width, height)
{
  // Save image details to be uploaded to device memory
  dev_data_.ref_img = ref_img_.dev_ptr;
  dev_data_.curr_img = curr_img_.dev_ptr;
  dev_data_.sum_templ = sum_templ_.dev_ptr;
  dev_data_.const_templ_denom = const_templ_denom_.dev_ptr;
  dev_data_.mu = mu_.dev_ptr;
  dev_data_.sigma = sigma_.dev_ptr;
  dev_data_.a = a_.dev_ptr;
  dev_data_.b = b_.dev_ptr;
  dev_data_.convergence = convergence_.dev_ptr;
  dev_data_.epipolar_matches = epipolar_matches_.dev_ptr;

  // Save camera parameters
  dev_data_.cam = cam;
  dev_data_.one_pix_angle = cam.getOnePixAngle();
  dev_data_.width  = width;
  dev_data_.height = height;

  // Kernel configuration
  dim_block_.x = 16;
  dim_block_.y = 16;
  dim_grid_.x = (width  + dim_block_.x - 1) / dim_block_.x;
  dim_grid_.y = (height + dim_block_.y - 1) / dim_block_.y;
}

bool rmd::SeedMatrix::setReferenceImage(
    float *host_ref_img_align_row_maj,
    const SE3<float> &T_curr_world,
    const float &min_depth,
    const float &max_depth)
{
  // Upload reference image to device memory
  ref_img_.setDevData(host_ref_img_align_row_maj);
  // Set scene parameters
  dev_data_.scene.min_depth    = min_depth;
  dev_data_.scene.max_depth    = max_depth;
  dev_data_.scene.avg_depth    = (min_depth+max_depth)/2.0f;
  dev_data_.scene.depth_range  = max_depth - min_depth;
  dev_data_.scene.sigma_sq_max = dev_data_.scene.depth_range * dev_data_.scene.depth_range / 36.0f;
  // Algorithm parameters
  dev_data_.eta_inlier  = 0.7f;
  dev_data_.eta_outlier = 0.05f;
  dev_data_.epsilon     = dev_data_.scene.depth_range / 10000.0f;
  // Copy data to device memory
  dev_data_.setDevData();

  T_world_ref_ = T_curr_world.inv();

  rmd::bindTexture(ref_img_tex, ref_img_);

  rmd::seedInitKernel<<<dim_grid_, dim_block_>>>(dev_data_.dev_ptr);
  hipDeviceSynchronize();
  return true;
}

bool rmd::SeedMatrix::update(
    float *host_curr_img_align_row_maj,
    const SE3<float> &T_curr_world)
{
  const rmd::SE3<float> T_curr_ref = T_curr_world * T_world_ref_;

  // Upload current image to device memory
  curr_img_.setDevData(host_curr_img_align_row_maj);
  // Bind texture memory for the current image
  rmd::bindTexture(curr_img_tex, curr_img_);

  // ... and model parameters
  rmd::bindTexture(mu_tex, mu_);
  rmd::bindTexture(sigma_tex, sigma_);
  rmd::bindTexture(a_tex, a_);
  rmd::bindTexture(b_tex, b_);

  // Assest current convergence status
  rmd::seedCheckKernel<<<dim_grid_, dim_block_>>>(dev_data_.dev_ptr);
  hipDeviceSynchronize();
  rmd::bindTexture(convergence_tex, convergence_);

  // Establish epipolar correspondences
  // call epipolar matching kernel
  rmd::seedEpipolarMatch<<<dim_grid_, dim_block_>>>(
                                                    dev_data_.dev_ptr,
                                                    T_curr_ref);
  hipDeviceSynchronize();
  rmd::bindTexture(epipolar_matches_tex, epipolar_matches_);

  rmd::seedUpdateKernel<<<dim_grid_, dim_block_>>>(
                                                   dev_data_.dev_ptr,
                                                   T_curr_ref.inv());
  hipDeviceSynchronize();
  return true;
}

void rmd::SeedMatrix::downloadDepthmap(float *host_depthmap_align_row_maj) const
{
  mu_.getDevData(host_depthmap_align_row_maj);
}

#if RMD_DEBUG
void rmd::SeedMatrix::downloadSigmaSq(float *host_align_row_maj) const
{
  sigma_.getDevData(host_align_row_maj);
}
void rmd::SeedMatrix::downloadA(float *host_align_row_maj) const
{
  a_.getDevData(host_align_row_maj);
}
void rmd::SeedMatrix::downloadB(float *host_align_row_maj) const
{
  b_.getDevData(host_align_row_maj);
}
void rmd::SeedMatrix::downloadSumTempl(float *host_align_row_maj) const
{
  sum_templ_.getDevData(host_align_row_maj);
}
void rmd::SeedMatrix::downloadConstTemplDenom(float *host_align_row_maj) const
{
  const_templ_denom_.getDevData(host_align_row_maj);
}
#endif
