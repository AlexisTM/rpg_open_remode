#include <rmd/seed_matrix.cuh>
#include <rmd/texture_memory.cuh>

#include "seed_init.cu"

rmd::SeedMatrix::SeedMatrix(
    const size_t &width,
    const size_t &height,
    const PinholeCamera &cam)
  : m_width(width)
  , m_height(height)
{
  m_ref_img  = new PaddedMemory(width, height);
  m_ref_img->getDevData(m_host_data.ref_img);
  m_curr_img = new PaddedMemory(width, height);
  m_curr_img->getDevData(m_host_data.curr_img);

  m_mu = new PaddedMemory(width, height);
  m_mu->getDevData(m_host_data.mu);
  m_sigma = new PaddedMemory(width, height);
  m_sigma->getDevData(m_host_data.sigma);
  m_a = new PaddedMemory(width, height);
  m_a->getDevData(m_host_data.a);
  m_b = new PaddedMemory(width, height);
  m_b->getDevData(m_host_data.b);

  m_host_data.cam    = cam;
  m_host_data.one_pix_angle = cam.getOnePixAngle();
  m_host_data.width  = width;
  m_host_data.height = height;

  hipMalloc(&m_dev_ptr, sizeof(m_host_data));

  m_dim_block.x = 16;
  m_dim_block.y = 16;
  m_dim_grid.x = (width  + m_dim_block.x - 1) / m_dim_block.x;
  m_dim_grid.y = (height + m_dim_block.y - 1) / m_dim_block.y;
}

rmd::SeedMatrix::~SeedMatrix()
{
  delete m_ref_img;
  delete m_curr_img;
  delete m_mu;
  delete m_sigma;
  delete m_a;
  delete m_b;
  hipFree(m_dev_ptr);
}

bool rmd::SeedMatrix::setReferenceImage(
    float *host_ref_img_align_row_maj,
    const SE3<float> &T_curr_world,
    const float &min_depth,
    const float &max_depth)
{
  if(hipSuccess != hipMemcpy2D(
        m_host_data.ref_img.data,
        m_host_data.ref_img.pitch,
        host_ref_img_align_row_maj,
        m_width*sizeof(float),
        m_width*sizeof(float),
        m_height,
        hipMemcpyHostToDevice))
    return false;

  m_host_data.scene.min_depth = min_depth;
  m_host_data.scene.max_depth = max_depth;
  m_host_data.scene.avg_depth = (min_depth+max_depth)/2.0f;
  m_host_data.scene.depth_range = max_depth - min_depth;
  m_host_data.scene.sigma_sq_max = m_host_data.scene.depth_range * m_host_data.scene.depth_range / 36.0f;

  hipMemcpy(m_dev_ptr, &m_host_data, sizeof(m_host_data), hipMemcpyHostToDevice);

  m_T_world_ref = T_curr_world.inv();

  if(hipSuccess != rmd::bindTexture(ref_img_tex, *m_ref_img))
    return false;

  rmd::seedInitKernel<<<m_dim_grid, m_dim_block>>>(m_dev_ptr);

  return true;
}

bool rmd::SeedMatrix::update(
    float *host_curr_img_align_row_maj,
    const SE3<float> &T_curr_world)
{
  if(hipSuccess != hipMemcpy2D(
       m_host_data.curr_img.data,
       m_host_data.curr_img.pitch,
       host_curr_img_align_row_maj,
       m_width*sizeof(float),
       m_width*sizeof(float),
       m_height,
       hipMemcpyHostToDevice))
    return false;
  const rmd::SE3<float> T_curr_ref = T_curr_world * m_T_world_ref;

  if(hipSuccess != rmd::bindTexture(curr_img_tex, *m_curr_img))
    return false;
  if(hipSuccess != rmd::bindTexture(mu_tex, *m_mu))
    return false;
  if(hipSuccess != rmd::bindTexture(sigma_tex, *m_sigma))
    return false;
  if(hipSuccess != rmd::bindTexture(a_tex, *m_a))
    return false;
  if(hipSuccess != rmd::bindTexture(b_tex, *m_b))
    return false;

  return true;
}
