#include "hip/hip_runtime.h"
// This file is part of REMODE - REgularized MOnocular Depth Estimation.
//
// Copyright (C) 2014 Matia Pizzoli <matia dot pizzoli at gmail dot com>
// Robotics and Perception Group, University of Zurich, Switzerland
// http://rpg.ifi.uzh.ch
//
// REMODE is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// REMODE is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include <rmd/reduction.cuh>

namespace rmd
{

__global__
void countEqualKernel(
    int *out_dev_ptr,
    size_t out_stride,
    int *in_dev_ptr,
    size_t in_stride,
    size_t n,
    size_t m,
    int value)
{
  extern __shared__ int s_partial[];
  int count = 0;

  // Sum over the thread grid
  for(int x = blockIdx.x * blockDim.x + threadIdx.x;
      x < n;
      x += blockDim.x*gridDim.x)
  {
    for(int y = blockIdx.y * blockDim.y + threadIdx.y;
        y < m;
        y += blockDim.y*gridDim.y)
    {
      count += in_dev_ptr[y*in_stride+x];
      // if(value == in_dev_ptr[y*in_stride+x])
      //      {
      //       count += 1;
      //    }
      //in_dev_ptr[y*in_stride+x] += 128;
    }
  }
  s_partial[threadIdx.y*blockDim.x+threadIdx.x] = count;
  __syncthreads();

  // Sum over the intermediate result in shared memory
  for(int threads_x = blockDim.x >> 1;
      threads_x;
      threads_x >>= 1)
  {
    for(int threads_y = blockDim.y >> 1;
        threads_y;
        threads_y >>= 1)
    {
      if(threadIdx.x < threads_x && threadIdx.y < threads_y)
      {
        s_partial[threadIdx.y*blockDim.x+threadIdx.x] +=
            s_partial[(threadIdx.y+threads_y)*blockDim.x + threadIdx.x + threads_x];
      }
      __syncthreads();
    }
  }
  if((0 == threadIdx.x) && (0 == threadIdx.y))
  {
    out_dev_ptr[blockIdx.y*out_stride+blockIdx.x] = s_partial[0];
  }
}

} // rmd namespace

size_t rmd::countEqual(
    const rmd::DeviceImage<int> &in_img,
    int value)
{
  // Kernel configuration
  dim3 dim_block;
  dim3 dim_grid;
  dim_block.x = 16; // Num threads
  dim_block.y = 16;
  dim_grid.x = 4;   // Num blocks
  dim_grid.y = 4;
  // Compute required shared memory
  unsigned int sh_mem_size = dim_block.x * dim_block.y * sizeof(int);
  // Allocate intermediate result (TODO: this should be pre-allocated)
  int *d_partial;
  size_t d_partial_pitch;
  hipError_t err = hipMallocPitch(
        &d_partial,
        &d_partial_pitch,
        dim_grid.x*sizeof(int),
        dim_grid.y);
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to allocate device memory", err);
  const size_t d_partial_stride = d_partial_pitch / sizeof(int);
  // Allocate final result
  int *d_count;
  err = hipMalloc(&d_count, sizeof(int));
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to allocate device memory", err);

  countEqualKernel<<<dim_grid, dim_block, sh_mem_size>>>(d_partial,
                                                         d_partial_stride,
                                                         in_img.data,
                                                         in_img.stride,
                                                         in_img.width,
                                                         in_img.height,
                                                         value);
  countEqualKernel<<<1, dim_block, sh_mem_size>>>(d_count,
                                                  0,
                                                  d_partial,
                                                  d_partial_stride,
                                                  dim_grid.x,
                                                  dim_grid.y,
                                                  value);

  int h_count;
  err = hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to copy result from device to host", err);
  err = hipFree(d_count);
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to free device memory", err);
  err = hipFree(d_partial);
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to free device memory", err);
  return static_cast<size_t>(h_count);
}
