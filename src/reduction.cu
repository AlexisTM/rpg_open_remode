#include "hip/hip_runtime.h"
// This file is part of REMODE - REgularized MOnocular Depth Estimation.
//
// Copyright (C) 2014 Matia Pizzoli <matia dot pizzoli at gmail dot com>
// Robotics and Perception Group, University of Zurich, Switzerland
// http://rpg.ifi.uzh.ch
//
// REMODE is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// REMODE is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include <rmd/reduction.cuh>
#include <cuda_toolkit/helper_timer.h>

namespace rmd
{

__global__
void reductionSumKernel(int *out_dev_ptr,
                        size_t out_stride,
                        const int *in_dev_ptr,
                        size_t in_stride,
                        size_t n,
                        size_t m)
{
  extern __shared__ int s_partial[];
  int count = 0;

  // Sum over 2D thread grid, use (x,y) indices
  for(int x = blockIdx.x * blockDim.x + threadIdx.x;
      x < n;
      x += blockDim.x*gridDim.x)
  {
    for(int y = blockIdx.y * blockDim.y + threadIdx.y;
        y < m;
        y += blockDim.y*gridDim.y)
    {
      count += in_dev_ptr[y*in_stride+x];
    }
  }
  // Sums are written to shared memory, single index
  s_partial[threadIdx.y*blockDim.x+threadIdx.x] = count;
  __syncthreads();

  // Reduce over block sums stored in shared memory
  // Start using half the block threads,
  // halve the active threads at each iteration
  const int tid = threadIdx.y*blockDim.x+threadIdx.x;
  for (int num_active_threads = (blockDim.x*blockDim.y)>>1;
       num_active_threads;
       num_active_threads >>= 1 ) {
    if ( tid < num_active_threads)
    {
      s_partial[tid] += s_partial[tid+num_active_threads];
    }
    __syncthreads();
  }
  // Thread 0 writes the result for the block
  if(0 == tid)
  {
    out_dev_ptr[blockIdx.y*out_stride+blockIdx.x] = s_partial[0];
  }
}

__global__
void maskKernel(int *out_dev_ptr,
                size_t out_stride,
                const int *in_dev_ptr,
                size_t in_stride,
                size_t n,
                size_t m,
                int value)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x >= n || y >= m)
    return;

  out_dev_ptr[y*out_stride+x] = in_dev_ptr[y*in_stride+x] == value ? 1 : 0;
}

} // rmd namespace

// Sum image by reduction
// Cfr. listing 12.1 by N. Wilt, "The CUDA Handbook"
int rmd::sum(const int *in_img_data,
             size_t in_img_stride,
             size_t in_img_width,
             size_t in_img_height)
{
  // Kernel configuration
  dim3 dim_block;
  dim3 dim_grid;
  dim_block.x = 16; // Num threads
  dim_block.y = 16;
  dim_grid.x = 4;   // Num blocks
  dim_grid.y = 4;

  // Compute required amount of shared memory
  unsigned int sh_mem_size = dim_block.x * dim_block.y * sizeof(int);

  // Allocate intermediate result (TODO: this should be pre-allocated)
  int *d_partial;
  size_t d_partial_pitch;
  hipError_t err = hipMallocPitch(
        &d_partial,
        &d_partial_pitch,
        dim_grid.x*sizeof(int),
        dim_grid.y);
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to allocate device memory", err);
  const size_t d_partial_stride = d_partial_pitch / sizeof(int);

  // Allocate final result
  int *d_count;
  err = hipMalloc(&d_count, sizeof(int));
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to allocate device memory", err);

  reductionSumKernel<<<dim_grid, dim_block, sh_mem_size>>>(d_partial,
                                                           d_partial_stride,
                                                           in_img_data,
                                                           in_img_stride,
                                                           in_img_width,
                                                           in_img_height);

  reductionSumKernel<<<1, dim_block, sh_mem_size>>>(d_count,
                                                    0,
                                                    d_partial,
                                                    d_partial_stride,
                                                    dim_grid.x,
                                                    dim_grid.y);

  // download sum
  int h_count;
  err = hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
  if(hipSuccess != err)
    throw CudaException("sum: unable to copy result from device to host", err);

  // Free device memory
  err = hipFree(d_count);
  if(hipSuccess != err)
    throw CudaException("sum: unable to free device memory", err);
  err = hipFree(d_partial);
  if(hipSuccess != err)
    throw CudaException("sum: unable to free device memory", err);

  return h_count;
}

int rmd::sum(const rmd::DeviceImage<int> &in_img)
{
  return rmd::sum(in_img.data,
                  in_img.stride,
                  in_img.width,
                  in_img.height);
}

size_t rmd::countEqual(const DeviceImage<int> &in_img,
                       int value)
{
  // Allocate mask
  int *d_mask;
  size_t d_mask_pitch;
  hipError_t err = hipMallocPitch(
        &d_mask,
        &d_mask_pitch,
        in_img.width*sizeof(int),
        in_img.height);
  const size_t d_mask_stride = d_mask_pitch / sizeof(int);

  // Kernel configuration
  dim3 dim_block;
  dim3 dim_grid;
  dim_block.x = 16;
  dim_block.y = 16;
  dim_grid.x = (in_img.width  + dim_block.x - 1) / dim_block.x;
  dim_grid.y = (in_img.height + dim_block.y - 1) / dim_block.y;

  maskKernel<<<dim_grid, dim_block>>>(d_mask,
                                      d_mask_stride,
                                      in_img.data,
                                      in_img.stride,
                                      in_img.width,
                                      in_img.height,
                                      value);

  // Sum over mask
  int mask_sum = rmd::sum(d_mask,
                          d_mask_stride,
                          in_img.width,
                          in_img.height);

  // Free device memory
  err = hipFree(d_mask);
  if(hipSuccess != err)
    throw CudaException("countEqual: unable to free device memory", err);

  return static_cast<size_t>(mask_sum);
}
