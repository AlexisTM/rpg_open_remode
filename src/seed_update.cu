#include "hip/hip_runtime.h"
#ifndef RMD_SEED_UPDATE_CU
#define RMD_SEED_UPDATE_CU

#include <rmd/device_data.cuh>

namespace rmd
{

__global__
void seedUpdateKernel(DeviceData *dev_ptr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x >= dev_ptr->width || y >= dev_ptr->height)
    return;

  const float xx = x+0.5f;
  const float yy = y+0.5f;

  // Check convergence status of seed
  if(ConvergenceStates::UPDATING != tex2D(convergence_tex, xx, yy))
    return;

  // Retrieve current estimations of parameters
  const float mu = tex2D(mu_tex, xx, yy);
  const float sigma_sq = tex2D(sigma_tex, xx, yy);
  const float a = tex2D(a_tex, xx, yy);
  const float b = tex2D(b_tex, xx, yy);

  // Bayesian update
  const float2 px_ref = make_float2((float)x, (float)y);
  const float3 f_ref = normalize(dev_ptr->cam.cam2world(px_ref));
#if 0
  float2 px;
  bool projection_within_image = true;
  if (findEpipolarMatch<ZMSSD>(
        px,
        projection_within_image,
        px_ref,
        cam_curr.world2cam(T_curr_ref * (f_ref*mu)), /* = px_curr_mean */
        cam_curr.world2cam(T_curr_ref * (f_ref*fmaxf(mu - 3.0f*sqrtf(sigma_sq), 0.01f))), /* = px_curr_min_d */
        cam_curr.world2cam(T_curr_ref * (f_ref*(mu + (3.0f*sqrtf(sigma_sq))))), /* = px_curr_max_d  */
        patch_edge_size,
        width, height
        ))
  {
    // Seed to be updated x/y with epi match px
    out_epipolar_matches[y*stride_32f2 + x] = px;
    out_converged[c32s] = 0;
  }
  else if (projection_within_image)
  {
    out_converged[c32s] = -1;
  }
  else
  {
    out_converged[c32s] = -2;
  }
#endif
}

} // rmd namespace

#endif
