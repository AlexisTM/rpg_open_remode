#include "hip/hip_runtime.h"
#ifndef RMD_SEED_CHECK_CU
#define RMD_SEED_CHECK_CU

#include <rmd/device_data.cuh>
#include <rmd/texture_memory.cuh>
#include <rmd/seed_matrix.cuh>

namespace rmd
{

__global__
void seedCheckKernel(DeviceData *dev_ptr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x >= dev_ptr->width || y >= dev_ptr->height)
    return;

  if(x > dev_ptr->width-dev_ptr->patch.side-1 || y > dev_ptr->height-dev_ptr->patch.side-1 ||
     x < dev_ptr->patch.side || y < dev_ptr->patch.side)
  {
    dev_ptr->convergence->at(x, y) = ConvergenceStates::BORDER;
    return;
  }

  const float xx = x+0.5f;
  const float yy = y+0.5f;

  // Retrieve current estimations of parameters
  const float mu = tex2D(mu_tex, xx, yy);
  const float sigma_sq = tex2D(sigma_tex, xx, yy);
  const float a = tex2D(a_tex, xx, yy);
  const float b = tex2D(b_tex, xx, yy);

  // if E(inlier_ratio) > eta_inlier && sigma_sq < epsilon
  if( ((a / (a + b)) > dev_ptr->eta_inlier)
      && (sigma_sq < dev_ptr->epsilon) )
  { // The seed converged
    dev_ptr->convergence->at(x, y) = ConvergenceStates::CONVERGED;
  }
  else if((a-1) / (a + b - 2) < dev_ptr->eta_outlier)
  { // The seed failed to converge
    dev_ptr->convergence->at(x, y) = ConvergenceStates::DIVERGED;
  }
  else
  {
    dev_ptr->convergence->at(x, y) = ConvergenceStates::UPDATE;
  }
}

} // rmd namespace

#endif
